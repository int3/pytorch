#include <ATen/ATen.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/CUDAUtils.h>
#include <ATen/Dispatch.h>

#if defined(USE_ROCM) || defined(_MSC_VER) || (defined(CUDA_VERSION) && CUDA_VERSION < 11080)
#else
#include <hip/hip_runtime.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm_universal.h>
#include <cutlass/gemm/device/gemm_universal_streamk_with_broadcast.h>
#include <cutlass/epilogue/thread/activation.h>
#include <cutlass/epilogue/threadblock/fusion/visitors.hpp>
#include <cutlass/gemm/kernel/default_gemm_universal_with_visitor.h>

#define CUTLASS_STATUS_CHECK(status)                                    \
  {                                                                       \
    TORCH_CHECK(status == cutlass::Status::kSuccess,                      \
                "Got CUTLASS error: ", cutlassGetStatusString(status));   \
  }

namespace {
  enum class Activation{NONE, RELU, SILU};
}
#endif

namespace at {
namespace native {

#if defined(USE_ROCM) || defined(_MSC_VER) || (defined(CUDA_VERSION) && CUDA_VERSION < 11080)
#else
template<typename ElementInputA, typename ElementInputB, bool use_scale,
        bool use_bias, Activation activation = Activation::NONE>
Tensor mixed_dtypes_linear_cutlass(
    const Tensor& input, const Tensor& weight, const Tensor& scale,
    const Tensor& bias) {
  // Weight matrix is transposed implicitly, by considering that its
  // elements are given in column-major order in this method (the code
  // below still takes into accoun that it's not explicitly transposed
  // when inquring about its shape and strides.

  using LayoutInputA = cutlass::layout::RowMajor;
  using LayoutInputB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;

  const int length_m = input.size(0);
  const int length_k = input.size(1);
  const int length_n = weight.size(0);

  // Check for current CUTLASS limitations w.r.t. weight sizes.
  TORCH_CHECK(length_m % 16 == 0 && length_k % 16 == 0 && length_n % 16 == 0,
              "mixed_dtypes_linear_cutlass: Number of rows/columns of the "
              "operands must be divisible by ", 16);

  using ElementC = ElementInputA;
  using ElementScale = ElementInputA;
  using ElementBias = ElementInputA;
  using ElementAccumulator = float;
  using ElementEpilogue = float;
  using ElementOutput = ElementInputA;

  constexpr int AlignmentInputA = 128 / cutlass::sizeof_bits<ElementInputA>::value;
  constexpr int AlignmentInputB = 128 / cutlass::sizeof_bits<ElementInputB>::value;
  constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;
  constexpr int AlignmentScale = 128 / cutlass::sizeof_bits<ElementScale>::value;
  constexpr int AlignmentBias = 128 / cutlass::sizeof_bits<ElementBias>::value;
  constexpr int AlignmentOutput = 128 / cutlass::sizeof_bits<ElementOutput>::value;

  using SmArch = cutlass::arch::Sm80;
  using ThreadblockShape    = cutlass::gemm::GemmShape<128, 128, 64>;
  using WarpShape           = cutlass::gemm::GemmShape<64, 64, 32>;
  using InstructionShape    = cutlass::gemm::GemmShape<16, 8, 16>;
  using ThreadblockSwizzle = cutlass::gemm::threadblock::ThreadblockSwizzleStreamK;
  constexpr auto NumStages = 3;

  constexpr auto NumEVTEpilogueStages = 1;

  using ScaleTileThreadMap = cutlass::epilogue::threadblock::OutputTileThreadLayout<
      ThreadblockShape,
      WarpShape,
      ElementScale,
      AlignmentScale,
      NumEVTEpilogueStages
  >;
  using BiasTileThreadMap = cutlass::epilogue::threadblock::OutputTileThreadLayout<
      ThreadblockShape,
      WarpShape,
      ElementBias,
      AlignmentBias,
      NumEVTEpilogueStages
  >;
  using OutputTileThreadMap = cutlass::epilogue::threadblock::OutputTileThreadLayout<
      ThreadblockShape,
      WarpShape,
      ElementOutput,
      AlignmentOutput,
      NumEVTEpilogueStages
  >;

  using Accum = cutlass::epilogue::threadblock::VisitorAccFetch;

  using ScaleScalar =
      cutlass::epilogue::threadblock::VisitorScalarBroadcast<ElementScale>;
  using ScaleTensor =
      cutlass::epilogue::threadblock::VisitorAuxLoad<
          ScaleTileThreadMap,
          ElementScale,
          cute::Stride<int64_t, cute::_1, int64_t>>;
  using Scale = std::conditional_t<use_scale, ScaleTensor, ScaleScalar>;
  using ScaleArguments = typename Scale::Arguments;

  using ApplyScale = cutlass::epilogue::threadblock::VisitorCompute<
      cutlass::multiplies, ElementEpilogue, ElementEpilogue,
      cutlass::FloatRoundStyle::round_to_nearest
  >;
  using EVTApplyScale = cutlass::epilogue::threadblock::Sm80EVT<
      ApplyScale,
      Accum,
      Scale>;

  using BiasScalar =
      cutlass::epilogue::threadblock::VisitorScalarBroadcast<ElementBias>;
  using BiasTensor =
      cutlass::epilogue::threadblock::VisitorAuxLoad<
          BiasTileThreadMap,
          ElementBias,
          cute::Stride<int64_t, cute::_1, int64_t>>;
  using Bias = std::conditional_t<use_bias, BiasTensor, BiasScalar>;
  using BiasArguments = typename Bias::Arguments;

  using ApplyBias = cutlass::epilogue::threadblock::VisitorCompute<
      cutlass::plus, ElementEpilogue, ElementEpilogue,
      cutlass::FloatRoundStyle::round_to_nearest
  >;
  using EVTApplyBias = cutlass::epilogue::threadblock::Sm80EVT<
      ApplyBias,
      EVTApplyScale,
      Bias>;

  using ApplyActivationNone = cutlass::epilogue::threadblock::VisitorCompute<
      cutlass::epilogue::thread::Identity, ElementEpilogue, ElementEpilogue,
      cutlass::FloatRoundStyle::round_to_nearest
  >;
  using ApplyActivationReLu = cutlass::epilogue::threadblock::VisitorCompute<
      cutlass::epilogue::thread::ReLu, ElementEpilogue, ElementEpilogue,
      cutlass::FloatRoundStyle::round_to_nearest
  >;
  using ApplyActivationSiLu = cutlass::epilogue::threadblock::VisitorCompute<
      cutlass::epilogue::thread::SiLu, ElementEpilogue, ElementEpilogue,
      cutlass::FloatRoundStyle::round_to_nearest
  >;
  using ApplyActivation =
      std::conditional_t<
          activation == Activation::NONE,
          ApplyActivationNone,
          std::conditional_t<
              activation == Activation::RELU,
              ApplyActivationReLu,
              ApplyActivationSiLu>>;
  using EVTApplyActivation = cutlass::epilogue::threadblock::Sm80EVT<
      ApplyActivation,
      EVTApplyBias>;

  using Output = cutlass::epilogue::threadblock::VisitorAuxStore<
      OutputTileThreadMap, ElementOutput, cutlass::FloatRoundStyle::round_to_nearest,
      cute::Stride<int64_t, cute::_1, int64_t> // StrideMNL
  >;

  using EVTOutput = cutlass::epilogue::threadblock::Sm80EVT<
      Output,
      EVTApplyActivation>;

  using EVTKernel =
      typename cutlass::gemm::kernel::DefaultGemmWithVisitor<
      ElementInputA, LayoutInputA, cutlass::ComplexTransform::kNone, AlignmentInputA,
      ElementInputB, LayoutInputB, cutlass::ComplexTransform::kNone, AlignmentInputB,
      ElementC, LayoutC, AlignmentC,
      ElementAccumulator,
      ElementEpilogue,
      cutlass::arch::OpClassTensorOp,
      SmArch,
      ThreadblockShape,
      WarpShape,
      InstructionShape,
      EVTOutput,
      ThreadblockSwizzle,
      NumStages,
      cutlass::arch::OpMultiplyAddMixedInputUpcast,
      NumEVTEpilogueStages
  >::GemmKernel;

  using Gemm = cutlass::gemm::device::GemmUniversalBase<EVTKernel>;

  auto output = input.new_empty({length_m, length_n});

  cutlass::gemm::GemmCoord problem_size(length_m, length_n, length_k);
  constexpr auto SplitKFactor = 1;

  ScaleArguments scale_arguments{
    [&]() -> ScaleArguments {
      if constexpr (use_scale) {
        return {(ElementScale*)scale.data_ptr(),
                ElementScale(1),
                {cute::_0{}, cute::_1{}, problem_size.n()}};
      } else {
        return {ElementScale(1)};
      }
    }()
  };
  BiasArguments bias_arguments{
    [&]() -> BiasArguments {
      if constexpr (use_bias) {
        return {(ElementBias*)bias.data_ptr(),
                ElementBias(0),
                {cute::_0{}, cute::_1{}, problem_size.n()}};
      } else {
        return {ElementBias(0)};
      }
    }()
  };
  typename Output::Arguments output_arguments{
    (ElementOutput*)output.data_ptr(),
    {problem_size.n(), cute::_1{}, problem_size.mn().product()}
  };
  typename EVTOutput::Arguments callback_arguments{
    {
      {
        {
          {},                // Accum
          scale_arguments,   // Scale
          {}                 // ApplyScale
        },                   // EVTApplyScale
        bias_arguments,      // Bias
        {}                   // ApplyBias
      },                     // EVTApplyBias
      {}                     // ApplyActivation
    },                       // EVTApplyActivation
    output_arguments,        // Output
  };                         // EVTOutput
  constexpr auto AvailSms = -1;
  typename Gemm::Arguments arguments(
    cutlass::gemm::GemmUniversalMode::kGemm,
    problem_size,
    SplitKFactor,
    callback_arguments,                       // arguments of EVT callbacks
    (ElementInputA*)input.data_ptr(),
    (ElementInputB*)weight.data_ptr(),
    nullptr,                                  // ptr C (unused)
    nullptr,                                  // ptr D (unused)
    problem_size.mk().product(),              // batch stride A
    problem_size.nk().product(),              // batch stride B
    0,                                        // batch stride C (unused)
    0,                                        // batch stride D (unused)
    input.strides()[0],                       // stride A
    weight.strides()[0],                      // stride B
    0,                                        // stride C (unused)
    0,                                        // stride D (unused)
    AvailSms);

  Gemm gemm_op;

  cutlass::Status status;

  // Verify that GEMM operation with given arguments can be performed
  // by CUTLASS.
  status = gemm_op.can_implement(arguments);
  CUTLASS_STATUS_CHECK(status);

  // Allocate workspace for CUTLASS mixed datatypes GEMM kernel.
  const auto workspace_size = Gemm::get_workspace_size(arguments);
  auto workspace = input.new_empty({(int64_t)workspace_size},
                                  at::TensorOptions().dtype(at::kByte));

  // Initialize CUTLASS mixed datatypes GEMM object.
  status = gemm_op.initialize(arguments, workspace.data_ptr(),
                              at::cuda::getCurrentCUDAStream());
  CUTLASS_STATUS_CHECK(status);

  // Perform mixed datatypes GEMM operation.
  status = gemm_op.run(at::cuda::getCurrentCUDAStream());
  CUTLASS_STATUS_CHECK(status);

  C10_CUDA_KERNEL_LAUNCH_CHECK();

  return output;
}

template<typename ElementInputA, typename ElementInputB, bool use_scale,
        bool use_bias>
Tensor mixed_dtypes_linear_cutlass_dispatch_activation(
    const Tensor& input, const Tensor& weight, const Tensor& scale,
    const Tensor& bias, const c10::string_view& activation) {
  if (activation == "none") {
    return mixed_dtypes_linear_cutlass<
        ElementInputA, ElementInputB, use_scale, use_bias, Activation::NONE>(
            input, weight, scale, bias);
  } else if (activation == "relu") {
    return mixed_dtypes_linear_cutlass<
        ElementInputA, ElementInputB, use_scale, use_bias, Activation::RELU>(
            input, weight, scale, bias);
  } else if (activation == "silu") {
    return mixed_dtypes_linear_cutlass<
        ElementInputA, ElementInputB, use_scale, use_bias, Activation::SILU>(
            input, weight, scale, bias);
  }

  AT_ERROR("mixed_dtypes_linear_cutlass_dispatch_activation: Activation \"",
           activation, "\" is not supported");
  return Tensor{};
}

template<typename ElementInputA, typename ElementInputB>
Tensor mixed_dtypes_linear_cutlass_dispatch_scale_bias(
    const Tensor& input, const Tensor& weight, const Tensor& scale,
    const Tensor& bias, const c10::string_view& activation) {
    if (scale.numel() > 0) {
        if (bias.numel() > 0) {
            return mixed_dtypes_linear_cutlass_dispatch_activation<
                       ElementInputA,
                       ElementInputB,
                       true,
                       true>(input, weight, scale, bias, activation);
        }
        else {
            return mixed_dtypes_linear_cutlass_dispatch_activation<
                       ElementInputA,
                       ElementInputB,
                       true,
                       false>(input, weight, scale, bias, activation);
        }
    }
    else {
        if (bias.numel() > 0) {
            return mixed_dtypes_linear_cutlass_dispatch_activation<
                       ElementInputA,
                       ElementInputB,
                       false,
                       true>(input, weight, scale, bias, activation);
        }
        else {
            return mixed_dtypes_linear_cutlass_dispatch_activation<
                       ElementInputA,
                       ElementInputB,
                       false,
                       false>(input, weight, scale, bias, activation);
        }
    }
}
#endif

Tensor
_mixed_dtypes_linear2(const Tensor& input, const Tensor& weight,
                      const c10::optional<Tensor>& scale_opt,
                      const c10::optional<Tensor>& bias_opt,
                      const c10::optional<c10::string_view> activation_opt) {
#if defined(USE_ROCM) || defined(_MSC_VER) || (defined(CUDA_VERSION) && CUDA_VERSION < 11080)
  AT_ERROR("_mixed_dtypes_linear2: ROCm doesn't support CUTLASS");
  return Tensor{};
#else
  const auto scale = scale_opt.has_value() ? *scale_opt : Tensor{};
  const auto bias = bias_opt.has_value() ? *bias_opt : Tensor{};
  const auto activation = activation_opt.has_value() ? *activation_opt : "none";

  // For now, only CC 8.x devices are supported.
  const auto dprops = at::cuda::getCurrentDeviceProperties();
  const auto is_sm8x = dprops->major == 8;
  TORCH_CHECK(is_sm8x,
              "_mixed_dtypes_linear2: Supported only on GPUs with compute "
              "capability 8.x");

  // Validate datatypes of input tensors.
  TORCH_CHECK(input.dtype() == at::kHalf ||
              input.dtype() == at::kBFloat16,
              "_mixed_dtypes_linear2: The input datatype ", input.dtype(),
              " is not supported");
  TORCH_CHECK(weight.dtype() == at::kChar ||
              weight.dtype() == at::kByte,
              "_mixed_dtypes_linear2: The weight datatype ", weight.dtype(),
              " is not supported");
  if (bias.numel() != 0) {
    TORCH_CHECK(bias.dtype() == input.dtype(),
                "_mixed_dtypes_linear2: Expected bias datatype ", input.dtype(),
                " but got", bias.dtype());
  }
  if (scale.numel() != 0) {
    TORCH_CHECK(scale.dtype() == input.dtype(),
                "_mixed_dtypes_linear2: Expected scale datatype ",
                input.dtype(), " but got", scale.dtype());
  }

  // Squash the batch dimensions of the input tensor with its
  // next-to-last dimensions.
  const auto input_sizes = input.sizes().vec();
  const auto input_2d = input.reshape({-1, input_sizes.back()});

  // Validate layouts of input tensors.
  TORCH_CHECK(input_2d.layout() == Layout::Strided,
              "_mixed_dtypes_linear2: Expected input argument to be strided, "
              "but got layout ", input_2d.layout());
  TORCH_CHECK(input_2d.dim() == 2,
              "_mixed_dtypes_linear2: Expected input argument to be 2D tensor, "
              "got ", input_2d.dim(), " dims");
  const auto input_strides = input_2d.strides();
  TORCH_CHECK(input_strides[0] > 1 && input_strides[1] == 1,
              "_mixed_dtypes_linear2: Invalid strides for input argument: row "
              "stride = ", input_strides[0], ", column stride = ",
              input_strides[1]);
  TORCH_CHECK(weight.layout() == Layout::Strided,
              "_mixed_dtypes_linear2: Expected input argument to be strided, "
              "but got layout ", weight.layout());
  TORCH_CHECK(weight.dim() == 2,
              "_mixed_dtypes_linear2: Expected weight argument to be 2D "
              " tensor, got ", weight.dim(), " dims");
  const auto weight_strides = weight.strides();
  TORCH_CHECK(weight_strides[0] > 1 && weight_strides[1] == 1,
              "_mixed_dtypes_linear2: Invalid strides for weight argument: row "
              "stride = ", weight_strides[0], ", column stride = ",
              weight_strides[1]);
  if (scale.numel() != 0) {
    TORCH_CHECK(scale.layout() == Layout::Strided,
              "_mixed_dtypes_linear2: Expected scale argument to be strided, "
              "but got layout ", scale.layout());
    TORCH_CHECK(scale.dim() == 1,
                "_mixed_dtypes_linear: Expected scale argument to be 1D ",
                "tensor, got ", scale.dim(), " dims");
    const auto scale_strides = scale.strides();
    TORCH_CHECK(scale_strides[0] == 1,
              "_mixed_dtypes_linear2: Invalid strides for scale argument: "
              "element stride = ", scale_strides[0]);
  }
  if (bias.numel() != 0) {
    TORCH_CHECK(bias.layout() == Layout::Strided,
              "_mixed_dtypes_linear2: Expected bias argument to be strided, "
              "but got layout ", bias.layout());
    TORCH_CHECK(bias.dim() == 1,
                "_mixed_dtypes_linear: Expected bias argument to be 1D ",
                "tensor, got ", bias.dim(), " dims");
    const auto bias_strides = bias.strides();
    TORCH_CHECK(bias_strides[0] == 1,
              "_mixed_dtypes_linear2: Invalid strides for bias argument: "
              "element stride = ", bias_strides[0]);
  }

  // Validate sizes of input tensors.
  TORCH_CHECK(input_2d.size(1) == weight.size(1),
              "_mixed_dtypes_linear2: Expected input argument to have ",
              weight.size(1), " columns, but got ", input_2d.size(1));
  if (scale.numel() != 0) {
    TORCH_CHECK(scale.dim() == 1,
                "_mixed_dtypes_linear: Expected scale argument to have ",
                weight.size(0), " elements, got ", scale.numel(), " elements");
  }
  if (bias.numel() != 0) {
    TORCH_CHECK(bias.dim() == 1,
                "_mixed_dtypes_linear: Expected bias argument to have ",
                weight.size(0), " elements, got ", bias.numel(), " elements");
  }

  Tensor output;
  auto scalar_type_quant = weight.scalar_type();
  AT_DISPATCH_SWITCH(
      input.scalar_type(),
      "_mixed_dtypes_linear2",
      AT_DISPATCH_CASE(
          at::ScalarType::Half,
          [&]() {
            AT_DISPATCH_SWITCH(
                scalar_type_quant,
                "_mixed_dtypes_linear2",
                AT_DISPATCH_CASE(
                    at::ScalarType::Char,
                    [&]() {
                      output =
                          mixed_dtypes_linear_cutlass_dispatch_scale_bias<
                              cutlass::half_t,
                              int8_t>(input_2d, weight, scale, bias,
                                      activation);
                      return;
                    })
                AT_DISPATCH_CASE(
                    at::ScalarType::Byte,
                    [&]() {
                      output =
                          mixed_dtypes_linear_cutlass_dispatch_scale_bias<
                              cutlass::half_t,
                              uint8_t>(input_2d, weight, scale, bias,
                                       activation);
                      return;
                    }));
          })
      AT_DISPATCH_CASE(
          at::ScalarType::BFloat16,
          [&]() {
            AT_DISPATCH_SWITCH(
                scalar_type_quant,
                "_mixed_dtypes_linear2",
                AT_DISPATCH_CASE(
                    at::ScalarType::Char,
                    [&]() {
                      output =
                          mixed_dtypes_linear_cutlass_dispatch_scale_bias<
                              cutlass::bfloat16_t,
                              int8_t>(input_2d, weight, scale, bias,
                                      activation);
                      return;
                    })
                AT_DISPATCH_CASE(
                    at::ScalarType::Byte,
                    [&]() {
                      output =
                          mixed_dtypes_linear_cutlass_dispatch_scale_bias<
                              cutlass::bfloat16_t,
                              uint8_t>(input_2d, weight, scale, bias,
                                       activation);
                      return;
                    }));
          }));

  auto output_sizes = input_sizes;
  output_sizes.back() = weight.size(0);
  return output.reshape(output_sizes);
#endif
}

}  // namespace native
}  // namespace at
